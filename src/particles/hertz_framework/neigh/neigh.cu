#include "hip/hip_runtime.h"
/*
 * Neighbor list decomposition of the hertz pairwise kernel.
 */

#define NSLOT 96          //< maximum number of neighbors per particle
#define AOS_LAYOUT      //< AoS or SoA layout for read-only particle data
//#define MAP_BUILD_CHECK //< bounds and sanity checking in build_inverse_map
//#define NEWTON_THIRD    //< use Newton's third law to halve computation
//#define COMPUTE_TPA     //< thread-per-atom decomposition
#define PINNED_MEM      //< use pinned-memory for kernel output
//#define KERNEL_PRINT    //< debug printing in kernel
//#define DEBUG           //< add (i,j) index information to struct

#ifdef GPU_TIMER
  #include "cuda_timer.h"
#elif POSIX_TIMER
  #include "posix_timer.h"
#else
  #include "simple_timer.h"
#endif

#include "check_result_vector.h"
#include "cuda_common.h"
#include "framework.h"
#include "hertz_constants.h"
#include "inverse_map.h"
#include "pair_interaction.h"
#include "particle.h"
#include <sstream>

using namespace std;

// --------------------------------------------------------------------------
// DEVICE KERNEL
// --------------------------------------------------------------------------

__global__ void compute_kernel_tpa(
  int nparticles,
#ifdef AOS_LAYOUT
  struct particle *particle_aos,
#else
  struct particle particle_soa,
#endif
  int *numneigh,
#ifdef AOS_LAYOUT
  struct particle *neigh,
#else
  struct particle neigh,
#endif
  double3 *shear,
  double *force,
  double *torque
#ifdef NEWTON_THIRD
  ,
  double3 *fdelta,
  double3 *tdeltaj
#endif
  ) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nparticles && numneigh[idx] > 0) {
    double xi[3]; double xj[3];
    double vi[3]; double vj[3];
    double omegai[3]; double omegaj[3];
    double radiusi; double radiusj;
    double massi; double massj;
    int    typei; int typej;
    double shearij[3] = {0,0,0};
    double force_sum[3] = {0,0,0};
    double torque_sum[3] = {0,0,0};

#ifdef AOS_LAYOUT
    xi[0]     = particle_aos[idx].x[0];
    xi[1]     = particle_aos[idx].x[1];
    xi[2]     = particle_aos[idx].x[2];
    vi[0]     = particle_aos[idx].v[0];
    vi[1]     = particle_aos[idx].v[1];
    vi[2]     = particle_aos[idx].v[2];
    omegai[0] = particle_aos[idx].omega[0];
    omegai[1] = particle_aos[idx].omega[1];
    omegai[2] = particle_aos[idx].omega[2];
    radiusi   = particle_aos[idx].radius;
    massi     = particle_aos[idx].mass;
    typei     = particle_aos[idx].type;
#else
    xi[0]     = particle_soa.x[(idx*3)+0];
    xi[1]     = particle_soa.x[(idx*3)+1];
    xi[2]     = particle_soa.x[(idx*3)+2];
    vi[0]     = particle_soa.v[(idx*3)+0];
    vi[1]     = particle_soa.v[(idx*3)+1];
    vi[2]     = particle_soa.v[(idx*3)+2];
    omegai[0] = particle_soa.omega[(idx*3)+0];
    omegai[1] = particle_soa.omega[(idx*3)+1];
    omegai[2] = particle_soa.omega[(idx*3)+2];
    radiusi   = particle_soa.radius[idx];
    massi     = particle_soa.mass[idx];
    typei     = particle_soa.type[idx];
#endif

    for (int jj=0; jj<numneigh[idx]; jj++) {
      int neigh_idx = (idx*NSLOT)+jj;
#ifdef AOS_LAYOUT
      //int j   = neigh[neigh_idx].idx;
      xj[0]     = neigh[neigh_idx].x[0];
      xj[1]     = neigh[neigh_idx].x[1];
      xj[2]     = neigh[neigh_idx].x[2];
      vj[0]     = neigh[neigh_idx].v[0];
      vj[1]     = neigh[neigh_idx].v[1];
      vj[2]     = neigh[neigh_idx].v[2];
      omegaj[0] = neigh[neigh_idx].omega[0];
      omegaj[1] = neigh[neigh_idx].omega[1];
      omegaj[2] = neigh[neigh_idx].omega[2];
      radiusj   = neigh[neigh_idx].radius;
      massj     = neigh[neigh_idx].mass;
      typej     = neigh[neigh_idx].type;
#else
      //int j   = neigh.idx[neigh_idx];
      xj[0]     = neigh.x[(neigh_idx*3)+0];
      xj[1]     = neigh.x[(neigh_idx*3)+1];
      xj[2]     = neigh.x[(neigh_idx*3)+2];
      vj[0]     = neigh.v[(neigh_idx*3)+0];
      vj[1]     = neigh.v[(neigh_idx*3)+1];
      vj[2]     = neigh.v[(neigh_idx*3)+2];
      omegaj[0] = neigh.omega[(neigh_idx*3)+0];
      omegaj[1] = neigh.omega[(neigh_idx*3)+1];
      omegaj[2] = neigh.omega[(neigh_idx*3)+2];
      radiusj   = neigh.radius[neigh_idx];
      massj     = neigh.mass[neigh_idx];
      typej     = neigh.type[neigh_idx];
#endif

      shearij[0] = shear[neigh_idx].x;
      shearij[1] = shear[neigh_idx].y;
      shearij[2] = shear[neigh_idx].z;

      double force_deltai[3] = {0,0,0};
      double torque_deltai[3] = {0,0,0};

#ifdef NEWTON_THIRD
      double torque_deltaj[3] = {0,0,0};
      pair_interaction(
#ifdef DEBUG
        particle_aos[idx].idx, neigh[neigh_idx].idx,
#endif
        xi, xj,
        vi, vj,
        omegai, omegaj,
        radiusi, radiusj,
        massi, massj,
        typei, typej,
        shearij,
        force_deltai, /* forcej is */ NULL,
        torque_deltai, torque_deltaj);

      fdelta[neigh_idx].x = force_deltai[0];
      fdelta[neigh_idx].y = force_deltai[1];
      fdelta[neigh_idx].z = force_deltai[2];

      tdeltaj[neigh_idx].x = torque_deltaj[0];
      tdeltaj[neigh_idx].y = torque_deltaj[1];
      tdeltaj[neigh_idx].z = torque_deltaj[2];
#else
      pair_interaction(
#ifdef DEBUG
        particle_aos[idx].idx, neigh[neigh_idx].idx,
#endif
        xi, xj,
        vi, vj,
        omegai, omegaj,
        radiusi, radiusj,
        massi, massj,
        typei, typej,
        shearij,
        force_deltai,  /* forcej is */ NULL,
        torque_deltai, /* torquej is */ NULL);
#endif

      shear[neigh_idx].x = shearij[0];
      shear[neigh_idx].y = shearij[1];
      shear[neigh_idx].z = shearij[2];

      force_sum[0] += force_deltai[0];
      force_sum[1] += force_deltai[1];
      force_sum[2] += force_deltai[2];

      torque_sum[0] += torque_deltai[0];
      torque_sum[1] += torque_deltai[1];
      torque_sum[2] += torque_deltai[2];

      if (idx==10) {
        cuPrintf("%.16f, %.16f, %.16f\n", force_deltai[0], force_deltai[1], force_deltai[2]);
      } else if (neigh_idx==10) {
        cuPrintf("%.16f, %.16f, %.16f\n", -force_deltai[0], -force_deltai[1], -force_deltai[2]);
      }

    }
    force[(idx*3)  ] += force_sum[0];
    force[(idx*3)+1] += force_sum[1];
    force[(idx*3)+2] += force_sum[2];

    torque[(idx*3)  ] += torque_sum[0];
    torque[(idx*3)+1] += torque_sum[1];
    torque[(idx*3)+2] += torque_sum[2];
  }
}

//NB: expects blockDim.x >= NSLOT
//    this kernel will produce incorrect results if blockDim.x < NSLOT
__global__ void compute_kernel_bpa(
  int nparticles,
#ifdef AOS_LAYOUT
  struct particle *particle_aos,
#else
  struct particle particle_soa,
#endif
  int *numneigh,
#ifdef AOS_LAYOUT
  struct particle *neigh,
#else
  struct particle neigh,
#endif
  double3 *shear,
  double *force,
  double *torque
#ifdef NEWTON_THIRD
  ,
  double3 *fdelta,
  double3 *tdeltaj
#endif
  ) {

  __shared__ double ftmp[NSLOT*3];
  __shared__ double ttmp[NSLOT*3];

  int jj = threadIdx.x;
  int idx = blockIdx.x;
  if (idx < nparticles && jj < numneigh[idx]) {
    double xi[3]; double xj[3];
    double vi[3]; double vj[3];
    double omegai[3]; double omegaj[3];
    double radiusi; double radiusj;
    double massi; double massj;
    int    typei; int typej;
    double shearij[3] = {0,0,0};
    double force_deltai[3] = {0,0,0};
    double torque_deltai[3] = {0,0,0};

#ifdef AOS_LAYOUT
    xi[0]     = particle_aos[idx].x[0];
    xi[1]     = particle_aos[idx].x[1];
    xi[2]     = particle_aos[idx].x[2];
    vi[0]     = particle_aos[idx].v[0];
    vi[1]     = particle_aos[idx].v[1];
    vi[2]     = particle_aos[idx].v[2];
    omegai[0] = particle_aos[idx].omega[0];
    omegai[1] = particle_aos[idx].omega[1];
    omegai[2] = particle_aos[idx].omega[2];
    radiusi   = particle_aos[idx].radius;
    massi     = particle_aos[idx].mass;
    typei     = particle_aos[idx].type;
#else
    xi[0]     = particle_soa.x[(idx*3)+0];
    xi[1]     = particle_soa.x[(idx*3)+1];
    xi[2]     = particle_soa.x[(idx*3)+2];
    vi[0]     = particle_soa.v[(idx*3)+0];
    vi[1]     = particle_soa.v[(idx*3)+1];
    vi[2]     = particle_soa.v[(idx*3)+2];
    omegai[0] = particle_soa.omega[(idx*3)+0];
    omegai[1] = particle_soa.omega[(idx*3)+1];
    omegai[2] = particle_soa.omega[(idx*3)+2];
    radiusi   = particle_soa.radius[idx];
    massi     = particle_soa.mass[idx];
    typei     = particle_soa.type[idx];
#endif

    int neigh_idx = (idx*NSLOT)+jj;
#ifdef AOS_LAYOUT
    //int j   = neigh[neigh_idx].idx;
    xj[0]     = neigh[neigh_idx].x[0];
    xj[1]     = neigh[neigh_idx].x[1];
    xj[2]     = neigh[neigh_idx].x[2];
    vj[0]     = neigh[neigh_idx].v[0];
    vj[1]     = neigh[neigh_idx].v[1];
    vj[2]     = neigh[neigh_idx].v[2];
    omegaj[0] = neigh[neigh_idx].omega[0];
    omegaj[1] = neigh[neigh_idx].omega[1];
    omegaj[2] = neigh[neigh_idx].omega[2];
    radiusj   = neigh[neigh_idx].radius;
    massj     = neigh[neigh_idx].mass;
    typej     = neigh[neigh_idx].type;
#else
    //int j   = neigh.idx[neigh_idx];
    xj[0]     = neigh.x[(neigh_idx*3)+0];
    xj[1]     = neigh.x[(neigh_idx*3)+1];
    xj[2]     = neigh.x[(neigh_idx*3)+2];
    vj[0]     = neigh.v[(neigh_idx*3)+0];
    vj[1]     = neigh.v[(neigh_idx*3)+1];
    vj[2]     = neigh.v[(neigh_idx*3)+2];
    omegaj[0] = neigh.omega[(neigh_idx*3)+0];
    omegaj[1] = neigh.omega[(neigh_idx*3)+1];
    omegaj[2] = neigh.omega[(neigh_idx*3)+2];
    radiusj   = neigh.radius[neigh_idx];
    massj     = neigh.mass[neigh_idx];
    typej     = neigh.type[neigh_idx];
#endif

    shearij[0] = shear[neigh_idx].x;
    shearij[1] = shear[neigh_idx].y;
    shearij[2] = shear[neigh_idx].z;

#ifdef NEWTON_THIRD
    double torque_deltaj[3] = {0,0,0};
    pair_interaction(
#ifdef DEBUG
      particle_aos[idx].idx, neigh[neigh_idx].idx,
#endif
      xi, xj,
      vi, vj,
      omegai, omegaj,
      radiusi, radiusj,
      massi, massj,
      typei, typej,
      shearij,
      force_deltai, /* forcej is */ NULL,
      torque_deltai, torque_deltaj);

    fdelta[neigh_idx].x = force_deltai[0];
    fdelta[neigh_idx].y = force_deltai[1];
    fdelta[neigh_idx].z = force_deltai[2];

    tdeltaj[neigh_idx].x = torque_deltaj[0];
    tdeltaj[neigh_idx].y = torque_deltaj[1];
    tdeltaj[neigh_idx].z = torque_deltaj[2];
#else
    pair_interaction(
#ifdef DEBUG
      particle_aos[idx].idx, neigh[neigh_idx].idx,
#endif
      xi, xj,
      vi, vj,
      omegai, omegaj,
      radiusi, radiusj,
      massi, massj,
      typei, typej,
      shearij,
      force_deltai,  /* forcej is */ NULL,
      torque_deltai, /* torquej is */ NULL);
#endif

    shear[neigh_idx].x = shearij[0];
    shear[neigh_idx].y = shearij[1];
    shear[neigh_idx].z = shearij[2];

    ftmp[(jj*3)  ] = force_deltai[0];
    ftmp[(jj*3)+1] = force_deltai[1];
    ftmp[(jj*3)+2] = force_deltai[2];

    ttmp[(jj*3)  ] = torque_deltai[0];
    ttmp[(jj*3)+1] = torque_deltai[1];
    ttmp[(jj*3)+2] = torque_deltai[2];

    __syncthreads();

    //reduce
    if (jj == 0) {
      //serial reduce
      for (int i=1; i<numneigh[idx]; i++) {
        force_deltai[0] += ftmp[(i*3)  ];
        force_deltai[1] += ftmp[(i*3)+1];
        force_deltai[2] += ftmp[(i*3)+2];

        torque_deltai[0] += ttmp[(i*3)  ];
        torque_deltai[1] += ttmp[(i*3)+1];
        torque_deltai[2] += ttmp[(i*3)+2];
      }

      force[(idx*3)  ] += force_deltai[0];
      force[(idx*3)+1] += force_deltai[1];
      force[(idx*3)+2] += force_deltai[2];

      torque[(idx*3)  ] += torque_deltai[0];
      torque[(idx*3)+1] += torque_deltai[1];
      torque[(idx*3)+2] += torque_deltai[2];
    }
  }
}

__global__ void gather_kernel(
  int nparticles,
  double3 *force_delta, double3 *torquej_delta,
  int *joffset, int *jcount, int *jmapinv,
  //outputs
  double *force, double *torque) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nparticles) {
    double fdelta[3] = {0.0, 0.0, 0.0};
    double tdelta[3] = {0.0, 0.0, 0.0};

    int joff = joffset[idx];
    for (int i=0; i<jcount[idx]; i++) {
      int e = jmapinv[joff+i];

      fdelta[0] -= force_delta[e].x;
      fdelta[1] -= force_delta[e].y;
      fdelta[2] -= force_delta[e].z;

      tdelta[0] += torquej_delta[e].x;
      tdelta[1] += torquej_delta[e].y;
      tdelta[2] += torquej_delta[e].z;
    }

    //output
    force[(idx*3)]   += fdelta[0];
    force[(idx*3)+1] += fdelta[1];
    force[(idx*3)+2] += fdelta[2];

    torque[(idx*3)]   += tdelta[0];
    torque[(idx*3)+1] += tdelta[1];
    torque[(idx*3)+2] += tdelta[2];
  }
}

// --------------------------------------------------------------------------
// RUN
// --------------------------------------------------------------------------

void insert_particle(struct params *input, 
#ifdef AOS_LAYOUT
  struct particle *particle_aos, 
#else
  struct particle particle_soa, 
#endif
  int id, int n) {
  assert(n < input->nnode);
#ifdef AOS_LAYOUT
  particle_aos[id].idx      = n;
  particle_aos[id].x[0]     = input->x[(n*3)  ];
  particle_aos[id].x[1]     = input->x[(n*3)+1];
  particle_aos[id].x[2]     = input->x[(n*3)+2];
  particle_aos[id].v[0]     = input->v[(n*3)  ];
  particle_aos[id].v[1]     = input->v[(n*3)+1];
  particle_aos[id].v[2]     = input->v[(n*3)+2];
  particle_aos[id].omega[0] = input->omega[(n*3)  ];
  particle_aos[id].omega[1] = input->omega[(n*3)+1];
  particle_aos[id].omega[2] = input->omega[(n*3)+2];
  particle_aos[id].radius   = input->radius[n];
  particle_aos[id].mass     = input->mass[n];
  particle_aos[id].type     = input->type[n];
#else
  particle_soa.idx[id]         = n;
  particle_soa.x[(id*3)+0]     = input->x[(n*3)  ];
  particle_soa.x[(id*3)+1]     = input->x[(n*3)+1];
  particle_soa.x[(id*3)+2]     = input->x[(n*3)+2];
  particle_soa.v[(id*3)+0]     = input->v[(n*3)  ];
  particle_soa.v[(id*3)+1]     = input->v[(n*3)+1];
  particle_soa.v[(id*3)+2]     = input->v[(n*3)+2];
  particle_soa.omega[(id*3)+0] = input->omega[(n*3)  ];
  particle_soa.omega[(id*3)+1] = input->omega[(n*3)+1];
  particle_soa.omega[(id*3)+2] = input->omega[(n*3)+2];
  particle_soa.radius[id]      = input->radius[n];
  particle_soa.mass[id]        = input->mass[n];
  particle_soa.type[id]        = input->type[n];
#endif
}

void build_particle_list(struct params *input, 
#ifdef AOS_LAYOUT 
    struct particle *&d_particle_aos
#else
    struct particle &d_particle_soa
#endif 
) {
#ifdef AOS_LAYOUT
  struct particle *particle_aos = new particle[input->nnode];
  for (int n=0; n<input->nnode; n++) {
    insert_particle(input, particle_aos, n, n);
  }
  const int aos_size = input->nnode*sizeof(struct particle);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_aos, aos_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_aos, particle_aos, aos_size, hipMemcpyHostToDevice));
#else
  struct particle particle_soa;
  particle_soa.idx    = new int[input->nnode];
  particle_soa.x      = new double[input->nnode*3];
  particle_soa.v      = new double[input->nnode*3];
  particle_soa.omega  = new double[input->nnode*3];
  particle_soa.radius = new double[input->nnode];
  particle_soa.mass   = new double[input->nnode];
  particle_soa.type   = new int[input->nnode];
  for (int n=0; n<input->nnode; n++) {
    insert_particle(input, particle_soa, n, n);
  }
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.idx, input->nnode*sizeof(int)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.x, input->nnode*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.v, input->nnode*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.omega, input->nnode*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.radius, input->nnode*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.mass, input->nnode*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_particle_soa.type, input->nnode*sizeof(int)));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.idx, particle_soa.idx, input->nnode*sizeof(int), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.x, particle_soa.x, input->nnode*3*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.v, particle_soa.v, input->nnode*3*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.omega, particle_soa.omega, input->nnode*3*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.radius, particle_soa.radius, input->nnode*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.mass, particle_soa.mass, input->nnode*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_particle_soa.type, particle_soa.type, input->nnode*sizeof(int), hipMemcpyHostToDevice));
#endif
}

void build_neighbor_list(
  int nslot,
  struct params *input,
  int *&d_numneigh, 
#ifdef AOS_LAYOUT
  struct particle *&d_neigh,
#else
  struct particle &d_neigh,
#endif
  double3 *&d_shear
#ifdef NEWTON_THIRD
  ,
  int &delta_size, double3 *&d_fdelta, double3 *&d_tdeltaj,
  int *&d_joffset, int *&d_jcount, int *&d_jmapinv
#endif
  ) {

  //numneigh[n]
  //is the number of neighbors for particle n
  int *numneigh = new int[input->nnode*nslot];
  //neigh[(n*nslot)+i]
  //is the struct for the i-th particle in contact with particle n
#ifdef AOS_LAYOUT
  struct particle *neigh = new particle[input->nnode*nslot];
#else
  struct particle neigh;
  neigh.idx    = new int[input->nnode*nslot];
  neigh.x      = new double[input->nnode*nslot*3];
  neigh.v      = new double[input->nnode*nslot*3];
  neigh.omega  = new double[input->nnode*nslot*3];
  neigh.radius = new double[input->nnode*nslot];
  neigh.mass   = new double[input->nnode*nslot];
  neigh.type   = new int[input->nnode*nslot];
#endif
  //shear[(n*nslot)+i]
  //is the shear for the i-th particle in contact with particle n
  double3 *shear = new double3[input->nnode*nslot];

  for (int i=0; i<input->nnode*nslot; i++) {
    numneigh[i] = 0;
  }
  for (int e=0; e<input->nedge; e++) {
    int n1 = input->edge[(e*2)  ];
    int n2 = input->edge[(e*2)+1];

    assert(numneigh[n1] < nslot);
    int idx = (n1*nslot) + numneigh[n1];
    insert_particle(input, neigh, idx, n2);
    shear[idx].x = input->shear[(e*3)  ];
    shear[idx].y = input->shear[(e*3)+1];
    shear[idx].z = input->shear[(e*3)+2];
    numneigh[n1]++;

#ifndef NEWTON_THIRD
    //insert the symmetric contact if not using Newton's Third Law
    assert(numneigh[n2] < nslot);
    idx = (n2*nslot) + numneigh[n2];
    insert_particle(input, neigh, idx, n1);
    shear[idx].x = input->shear[(e*3)  ];
    shear[idx].y = input->shear[(e*3)+1];
    shear[idx].z = input->shear[(e*3)+2];
    numneigh[n2]++;
#endif
  }

  const int numneigh_size = input->nnode*nslot*sizeof(int);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_numneigh, numneigh_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_numneigh, numneigh, numneigh_size, hipMemcpyHostToDevice));

#ifdef AOS_LAYOUT
  const int neigh_size = input->nnode*nslot*sizeof(struct particle);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh, neigh_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh, neigh, neigh_size, hipMemcpyHostToDevice));
#else
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.idx, input->nnode*nslot*sizeof(int)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.x, input->nnode*nslot*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.v, input->nnode*nslot*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.omega, input->nnode*nslot*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.radius, input->nnode*nslot*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.mass, input->nnode*nslot*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_neigh.type, input->nnode*nslot*sizeof(int)));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.idx, neigh.idx, input->nnode*nslot*sizeof(int), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.x, neigh.x, input->nnode*3*nslot*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.v, neigh.v, input->nnode*3*nslot*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.omega, neigh.omega, input->nnode*3*nslot*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.radius, neigh.radius, input->nnode*nslot*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.mass, neigh.mass, input->nnode*nslot*sizeof(double), hipMemcpyHostToDevice));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_neigh.type, neigh.type, input->nnode*nslot*sizeof(int), hipMemcpyHostToDevice));
#endif

  const int shear_size = input->nnode*nslot*sizeof(double3);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_shear, shear_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_shear, shear, shear_size, hipMemcpyHostToDevice));

#ifdef NEWTON_THIRD
  //jmap[(n*nslot)+i]
  //is the idx of the i-th particle in contact with particle n
  int *jmap = new int[input->nnode*nslot];
  for (int i=0; i<input->nnode*nslot; i++) {
    jmap[i] = neigh[i].idx;
  }

  //build an inverse mapping of jmap
  int *joffset;
  int *jcount;
  int *jmapinv;
  build_inverse_map(
    numneigh, jmap, input->nnode/*T*/, nslot, input->nnode/*K*/,
    joffset, jcount, jmapinv);

  const int joffset_jcount_size = input->nnode*sizeof(int);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_joffset, joffset_jcount_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_joffset, joffset, joffset_jcount_size, hipMemcpyHostToDevice));

  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_jcount, joffset_jcount_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_jcount, jcount, joffset_jcount_size, hipMemcpyHostToDevice));

  const int jmapinv_size = input->nnode*nslot*sizeof(int);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_jmapinv, jmapinv_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(d_jmapinv, jmapinv, jmapinv_size, hipMemcpyHostToDevice));

  delta_size = input->nnode*nslot*sizeof(double3);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_fdelta, delta_size));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_tdeltaj, delta_size));
#endif

}

// --------------------------------------------------------------------------
// RUN
// --------------------------------------------------------------------------

void run(struct params *input, int num_iter) {

  //--------------------
  // One-time only costs
  //--------------------
  one_time.push_back(SimpleTimer("hertz_constants"));
  one_time.back().start();
  setup_hertz_constants(input);
  one_time.back().stop_and_add_to_total();

  one_time.push_back(SimpleTimer("build_particle_list"));
#ifdef AOS_LAYOUT
  struct particle *d_particle_list = NULL;
#else
  struct particle d_particle_list;
#endif
  one_time.back().start();
  build_particle_list(input, d_particle_list);
  one_time.back().stop_and_add_to_total();
#ifdef AOS_LAYOUT
  assert(d_particle_list);
#endif

  one_time.push_back(SimpleTimer("build_neigh_list"));
  int *d_numneigh = NULL;
#ifdef AOS_LAYOUT
  struct particle *d_neigh = NULL;
#else
  struct particle d_neigh;
#endif
  double3 *d_shear = NULL;
#ifdef NEWTON_THIRD
  int delta_size;
  double3 *d_fdelta = NULL;
  double3 *d_tdeltaj = NULL;
  int *d_joffset = NULL;
  int *d_jcount = NULL;
  int *d_jmapinv = NULL;
  one_time.back().start();
  build_neighbor_list(NSLOT, input, d_numneigh, d_neigh, d_shear,
    delta_size, d_fdelta, d_tdeltaj,
    d_joffset, d_jcount, d_jmapinv);
  one_time.back().stop_and_add_to_total();
  assert(d_fdelta);
  assert(d_tdeltaj);
  assert(d_joffset);
  assert(d_jcount);
  assert(d_jmapinv);
#else
  one_time.back().start();
  build_neighbor_list(NSLOT, input, d_numneigh, d_neigh, d_shear);
  one_time.back().stop_and_add_to_total();
#endif
  assert(d_numneigh);
#ifdef AOS_LAYOUT
  assert(d_neigh);
#endif
  assert(d_shear);

  one_time.push_back(SimpleTimer("malloc_force_torque"));
  one_time.back().start();
  double *d_force;
  const int force_size = input->nnode * 3 * sizeof(double);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_force, force_size));

  double *d_torque;
  const int torque_size = input->nnode * 3 * sizeof(double);
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_torque, torque_size));
  one_time.back().stop_and_add_to_total();

  //TODO(1): copy real x, v, omega in PREPROCESS
  //These are dummy structures just for timing
  const int d_x_size = input->nnode * 3 * sizeof(double);
  double *d_fake_x;
  double *d_fake_v;
  double *d_fake_omega;
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_fake_x, d_x_size));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_fake_v, d_x_size));
  ASSERT_NO_CUDA_ERROR(
    hipMalloc((void **)&d_fake_omega, d_x_size));

#ifdef PINNED_MEM
  one_time.push_back(SimpleTimer("pinned_mem"));
  one_time.back().start();
  double *h_x;
  double *h_v;
  double *h_omega;
  double *h_force;
  double *h_torque;
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&h_x, d_x_size));
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&h_v, d_x_size));
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&h_omega, d_x_size));
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&h_force, force_size));
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&h_torque, torque_size));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(h_x, input->x, d_x_size, hipMemcpyHostToHost));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(h_v, input->v, d_x_size, hipMemcpyHostToHost));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(h_omega, input->omega, d_x_size, hipMemcpyHostToHost));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(h_force, input->force, force_size, hipMemcpyHostToHost));
  ASSERT_NO_CUDA_ERROR(
    hipMemcpy(h_torque, input->torque, torque_size, hipMemcpyHostToHost));
  one_time.back().stop_and_add_to_total();
#endif

  //--------------------
  // Per-iteration costs
  //--------------------

  per_iter.push_back(SimpleTimer("memcpy_to_dev"));
#ifdef COMPUTE_TPA
  per_iter.push_back(SimpleTimer("compute_kernel_tpa"));
#else
  per_iter.push_back(SimpleTimer("compute_kernel_bpa"));
#endif
  per_iter.push_back(SimpleTimer("gather_kernel"));
  per_iter.push_back(SimpleTimer("result_fetch"));

#ifdef PINNED_MEM
  double3 *shear_result;
  double *force_result;
  double *torque_result;
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&shear_result, input->nnode*NSLOT*sizeof(double3)));
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&force_result, input->nnode*3*sizeof(double)));
  ASSERT_NO_CUDA_ERROR(
    hipHostMalloc((void **)&torque_result, input->nnode*3*sizeof(double)));
#else
  double3 *shear_result = new double3[input->nnode*NSLOT];
  double *force_result = new double[input->nnode*3];
  double *torque_result = new double[input->nnode*3];
#endif

  for (int run=0; run<num_iter; run++) {
    //PREPROCESSING
    //copy across structures that change between kernel invocations,
    //reset delta structures (force/torque)
    //TODO(1): just copy dummy structures for timing
    per_iter[0].start();
#ifdef PINNED_MEM
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_fake_x, h_x, d_x_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_fake_v, h_v, d_x_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_fake_omega, h_omega, d_x_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_force, h_force, force_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_torque, h_torque, torque_size, hipMemcpyHostToDevice));
#else
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_fake_x, input->x, d_x_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_fake_v, input->v, d_x_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_fake_omega, input->omega, d_x_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_force, input->force, force_size, hipMemcpyHostToDevice));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(d_torque, input->torque, torque_size, hipMemcpyHostToDevice));
#endif
#ifdef NEWTON_THIRD
    ASSERT_NO_CUDA_ERROR(
      hipMemset((void *)d_fdelta, 0, delta_size));
    ASSERT_NO_CUDA_ERROR(
      hipMemset((void *)d_tdeltaj, 0, delta_size));
#endif
    per_iter[0].stop_and_add_to_total();

    //-----------------------------------------------------------------------

    //KERNEL INVOCATION
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Pre-compute-kernel error: %s.\n", hipGetErrorString(err));
      exit(1);
    }

#ifdef KERNEL_PRINT
    cudaPrintfInit();
#endif
    per_iter[1].start();
#ifdef NEWTON_THIRD
  #ifdef COMPUTE_TPA
    const int blockSize = 128;
    dim3 gridSize((input->nnode / blockSize)+1);
    compute_kernel_tpa<<<gridSize, blockSize>>>(
      input->nnode, d_particle_list, d_numneigh, d_neigh,
      d_shear, d_force, d_torque, d_fdelta, d_tdeltaj);
  #else //COMPUTE_BPA
    const int blockSize = NSLOT;
    dim3 gridSize(input->nnode);
    compute_kernel_bpa<<<gridSize, blockSize>>>(
      input->nnode, d_particle_list, d_numneigh, d_neigh,
      d_shear, d_force, d_torque, d_fdelta, d_tdeltaj);
  #endif
#else
  #ifdef COMPUTE_TPA
    const int blockSize = 128;
    dim3 gridSize((input->nnode / blockSize)+1);
    compute_kernel_tpa<<<gridSize, blockSize>>>(
      input->nnode, d_particle_list, d_numneigh, d_neigh,
      d_shear, d_force, d_torque);
  #else //COMPUTE_BPA
    const int blockSize = NSLOT;
    dim3 gridSize(input->nnode);
    compute_kernel_bpa<<<gridSize, blockSize>>>(
      input->nnode, d_particle_list, d_numneigh, d_neigh,
      d_shear, d_force, d_torque);
  #endif
#endif
    hipDeviceSynchronize();
    per_iter[1].stop_and_add_to_total();

    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Post-compute-kernel error: %s.\n", hipGetErrorString(err));
      exit(1);
    }

#ifdef NEWTON_THIRD
    const int gatherBlockSize = 128;
    dim3 gatherGridSize((input->nnode / gatherBlockSize)+1);
    per_iter[2].start();
    gather_kernel<<<gatherGridSize, gatherBlockSize>>>(
      input->nnode,
      d_fdelta, d_tdeltaj,
      d_joffset, d_jcount, d_jmapinv,
      d_force, d_torque);
    hipDeviceSynchronize();
    per_iter[2].stop_and_add_to_total();

    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Post-gather error: %s.\n", hipGetErrorString(err));
      exit(1);
    }
#endif

#ifdef KERNEL_PRINT
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif

    //-----------------------------------------------------------------------

    //POSTPROCESSING
    //memcpy data back to host
    const int shear_size = input->nnode*NSLOT*sizeof(double3);
    per_iter[3].start();
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(shear_result, d_shear, shear_size, hipMemcpyDeviceToHost));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(force_result, d_force, force_size, hipMemcpyDeviceToHost));
    ASSERT_NO_CUDA_ERROR(
      hipMemcpy(torque_result, d_torque, torque_size, hipMemcpyDeviceToHost));
    per_iter[3].stop_and_add_to_total();

#if 0
    if (run == 0) {
      for (int n=0; n<input->nnode; n++) {
        std::stringstream out;
        out << "force[" << n << "]";
        check_result_vector(
            out.str().c_str(),
            &input->expected_force[(n*3)], &force_result[(n*3)]);
        out.str("");

        out << "torque[" << n << "]";
        check_result_vector(
            out.str().c_str(),
            &input->expected_torque[(n*3)], &torque_result[(n*3)]);
      }
      //TODO check shear
    }
#endif
  }

#ifdef PINNED_MEM
  hipHostFree(h_x);
  hipHostFree(h_v);
  hipHostFree(h_omega);
  hipHostFree(h_force);
  hipHostFree(h_torque);
  hipHostFree(shear_result);
  hipHostFree(force_result);
  hipHostFree(torque_result);
#endif
#ifdef NEWTON_THIRD
  hipFree(d_joffset);
  hipFree(d_jcount);
  hipFree(d_jmapinv);
  hipFree(d_fdelta);
  hipFree(d_tdeltaj);
#endif
#ifdef AOS_LAYOUT
  hipFree(d_particle_list);
#else
  hipFree(d_particle_list.idx);
  hipFree(d_particle_list.x);
  hipFree(d_particle_list.v);
  hipFree(d_particle_list.omega);
  hipFree(d_particle_list.radius);
  hipFree(d_particle_list.mass);
  hipFree(d_particle_list.type);
#endif
  hipFree(d_numneigh);
#ifdef AOS_LAYOUT
  hipFree(d_neigh);
#else
  hipFree(d_neigh.idx);
  hipFree(d_neigh.x);
  hipFree(d_neigh.v);
  hipFree(d_neigh.omega);
  hipFree(d_neigh.radius);
  hipFree(d_neigh.mass);
  hipFree(d_neigh.type);
#endif
  hipFree(d_shear);
  hipFree(d_force);
  hipFree(d_torque);
  hipFree(d_fake_x);
  hipFree(d_fake_v);
  hipFree(d_fake_omega);
}
